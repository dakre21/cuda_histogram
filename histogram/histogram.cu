/*
* Author      : David Akre
* Date        : 10/15/17
* Description : This program will take in a file which will have a
4096x4096 matrix of numbers and compute a histogram of the number
of occurances numbers 0-9 occur in that matrix and output it to 
a file given by the second argument.
* 
* Usage       : Run './histogram <input_file.txt> <output_file.txt>
*/

#include <stdio.h>
#include <iostream>
#include <unistd.h>
#include <hip/hip_runtime.h>

using namespace std;

#define BLOCK_SIZE   1
#define THREAD_SIZE  1
#define NUM_ELEMENTS 10

// Global declaration for FILEs
FILE* file_out;
FILE* file_in;

__global__ void calc_histogram(char* dbuff, int* dcount, int size) {

    int index = threadIdx.x + blockDim.x*blockIdx.x;

    for (int i = index; i < size; i++) {
        // Increment counter per occurances
        if (dbuff[i] == '0') {
            dcount[0] += 1;
        } else if (dbuff[i] == '1') {
            dcount[1] += 1;
        } else if (dbuff[i] == '2') {
            dcount[2] += 1;
        } else if (dbuff[i] == '3') {
            dcount[3] += 1;
        } else if (dbuff[i] == '4') {
            dcount[4] += 1;
        } else if (dbuff[i] == '5') {
            dcount[5] += 1;
        } else if (dbuff[i] == '6') {
            dcount[6] += 1;
        } else if (dbuff[i] == '7') {
            dcount[7] += 1;
        } else if (dbuff[i] == '8') {
            dcount[8] += 1;
        } else if (dbuff[i] == '9') {
            dcount[9] += 1;
        }
    }
}

void app_exit(int rc) {
    // Close files and exit
    fclose(file_in);
    fclose(file_out);
    exit(rc);
}

int verify_inputs(char* argv[]) {
    // Begin verification steps
    if (argv[1] == NULL || argv[2] == NULL) {
        fprintf(stderr, "Invalid number of inputs\n");
        return -1;
    }

    // Attempt to open the input file
    file_in = fopen(argv[1], "r");

    // Check if the file exists
    if (file_in == NULL) {
        fprintf(stderr, "Input file does not exist\n");
        return -1;
    }

    // Attempt to create the output file
    file_out = fopen(argv[2], "w+");

    // Check if the file exists
    if (file_out == NULL) {
        fprintf(stderr, "Failed to create output file\n");
        return -1;
    }

    return 0;
}

int create_histogram() {
    // Forward declarations
    int rc           = 0;
    int size         = 0;
    char* buff;
    char* dbuff;
    int count[NUM_ELEMENTS] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    int* dcount;
    
    // Read the size of the file
    fseek(file_in, 0, SEEK_END);
    size = ftell(file_in) + 1;
    rewind(file_in);

    // Malloc space for CUDA
    hipMalloc((void**)&dbuff, size);
    hipMalloc((void**)&dcount, 10);

    // Create heap space for buffer
    buff = reinterpret_cast<char*>(malloc(size*sizeof(char)));
    memset(buff, '\0', size);

    // Read file
    fread(buff, size, sizeof(char), file_in);

    hipMemcpy(dbuff, buff, size, hipMemcpyHostToDevice);
    hipMemcpy(dcount, count, NUM_ELEMENTS, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid(1, 1);

    calc_histogram<<<BLOCK_SIZE, THREAD_SIZE>>>(dbuff, dcount, size);

    hipMemcpy(buff, dbuff, size, hipMemcpyDeviceToHost);
    hipMemcpy(count, dcount, NUM_ELEMENTS, hipMemcpyDeviceToHost);

    hipFree(dbuff);
    hipFree(dcount);
    free(buff);

    return rc;
}

int main (int argc, char* argv[]) {
    // Forward declarations
    int rc = 0;
    
    // Verify inputs
    rc = verify_inputs(argv);
    if (rc != 0) {
        fprintf(stderr, "Failed to verify input arguments. Appropriate "\
            "usage: ./histogram <input_matrix_file> <output_histogram_file>\n");
        app_exit(rc);
    }

    // Compute histogram math
    rc = create_histogram();
    if (rc != 0) {
        fprintf(stderr, "Failed to parse and calculate the histogram from "\
            "the input matrix\n");
        app_exit(rc);
    }

    // Exit app
    app_exit(rc);
}
